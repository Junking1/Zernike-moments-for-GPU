#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cuda_zernike_moment.h"
#include <stdio.h>
#include "base_cuda.h"
#include "reduction.h"
#include "time_count.h"
//#include "cpu_zernike_moment.h"
#include <math.h>


__device__ double device_factorial(int n)
{
	double value = 1;
	for (int i = 1; i <= n; i++)
	{
		value = value * i;
	}
	return value;
}//*/

__device__ double device_radial_poly(double r, int n, int m)
{
	double rad = 0;
	double c;
	double den;

	//for(int s = 0;s<=(n-abs(m))/2; s++)   //是否加1 不一定
	for (int s = 0; s <= (n - m) / 2; s++)   //是否加1 不一定
	{
		//den =  device_factorial(s) * device_factorial((n+fabs((double)m))/2-s) * device_factorial((n-fabs((double)m))/2-s);
		den = device_factorial(s) * device_factorial((n + (double)m) / 2 - s) * device_factorial((n - (double)m) / 2 - s);
		//den =  c_factorial[s] * c_factorial[(n+m)/2-s] * c_factorial[(n-m)/2-s];

		c = pow((double)-1, (double)s) * device_factorial(n-s)/den;
		//c = pow((double)-1, (double)s) * c_factorial[n - s] / den;
		rad = rad + c * pow(r, n - 2 * s);
	}
	return rad;
}



__global__ void moment_map_kernel(int * d_p, double * d_real_map, double * d_image_map, moment_para para)
{
	const int tid = blockDim.x * blockIdx.x + threadIdx.x;   //表示总的

	int i, j;
	double x, y;
	double r, theta;
	double R;
	double cent_position;
	double rad;
	double real_temp;
	double image_temp;
	double p;

	R = para.pixel_size / 2;
	cent_position = R - 0.5;

	i = tid % para.pixel_size;   //表示列， 代表x
	j = tid / para.pixel_size;   //表示行， 代表y

	x = i - cent_position;
	y = cent_position - j;// j-cent_position;    // 注意此处 y需要翻转
	r = sqrt(x*x/2 + y*y/2) / R;
	//r = r / sqrt(2);
	theta = atan2(y, x);
	__syncthreads();
	if (r<1)
	{
		rad = device_radial_poly(r, para.n, para.m);

		real_temp = rad * cos(para.m*theta) * d_p[tid];
		image_temp = -rad * sin(para.m*theta) * d_p[tid];
	}
	else
	{
		real_temp = 0;
		image_temp = 0;
	}

	d_real_map[tid] = real_temp;
	d_image_map[tid] = image_temp; //real image 颠倒顺序结果正确？？？？？

}



extern "C" void calc_znk_moment(int * d_p, double * d_real_map, double * d_image_map, struct moment_para para)
{
	int thread_num = para.thread_num;
	int block_num = para.pixel_size * para.pixel_size / thread_num;

	moment_map_kernel <<< block_num, thread_num >>> (d_p, d_real_map, d_image_map, para);


	/*switch (para.mode)
	{
	case NO_CONSTANT:
		moment_map_kernel_noconstant << < block_num, thread_num >> > (d_p, d_real_map, d_image_map, para);
		hipDeviceSynchronize();
		break;
	case FAC_CONSTANT:
		moment_map_kernel_fac << < block_num, thread_num >> > (d_p, d_real_map, d_image_map, para);
		hipDeviceSynchronize();
		break;
	case NMS_CONSTANT:
		moment_map_kernel_nms << < block_num, thread_num >> > (d_p, d_real_map, d_image_map, para);
		hipDeviceSynchronize();
		break;
	}*/

	//hipDeviceSynchronize();
}
